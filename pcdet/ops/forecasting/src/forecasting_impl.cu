#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//#include "pcdet/ops/iou3d_nms/src/iou3d_cpu.h"
#include "iou3d_cpu.h"

// Using double instead of float doesn't increase
// accuracy as far as I have tested
//#define USE_DOUBLE

// check https://discuss.pytorch.org/t/using-at-parallel-for-in-a-custom-operator/82747/4
// if you want to make cpu op parallel

#ifdef USE_DOUBLE
using fp_type = double;
#define FABS(x) fabs(x)
#define SQRT(x) sqrt(x)
#define COS(x) cos(x)
#define SIN(x) sin(x)
#define ATAN2(x,y) atan2(x,y)
#else
using fp_type = float;
#define FABS(x) fabsf(x)
#define SQRT(x) sqrtf(x)
#define COS(x) cosf(x)
#define SIN(x) sinf(x)
#define ATAN2(x,y) atan2f(x,y)
#endif

template <typename scalar_t>
using one_dim_pa32 = torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits>;

template <typename scalar_t>
using two_dim_pa32 = torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits>;

template <typename scalar_t>
using one_dim_acc = torch::TensorAccessor<scalar_t,1>;

template <typename scalar_t>
using two_dim_acc = torch::TensorAccessor<scalar_t,2>;

class Quaternion{
    public:
        __host__ __device__ Quaternion(fp_type r, fp_type i, fp_type j, fp_type k){
            r_ = r;
            i_ = i;
            j_ = j;
            k_ = k;
        }

        __host__ __device__ Quaternion(fp_type *axis, fp_type angle_rad){
            fp_type axis_x = axis[0];
            fp_type axis_y = axis[1];
            fp_type axis_z = axis[2];
            fp_type mag_sq = axis_x*axis_x + axis_y+axis_y + axis_z*axis_z;
            if(FABS(1.0f - mag_sq) > 1e-12){
                fp_type s = SQRT(mag_sq);
                axis_x /= s;
                axis_y /= s;
                axis_z /= s;
            }
            fp_type theta = angle_rad / 2.0;
            r_ = COS(theta);
            fp_type st = SIN(theta);
            i_ = axis_x * st;
            j_ = axis_y * st;
            k_ = axis_z * st;
        }

        __host__ __device__ void rmul_inplace(Quaternion &q){
            // multiply the q matrix of q with self
            fp_type r_tmp = q.r_*r_ - q.i_*i_ - q.j_*j_ - q.k_*k_;
            fp_type i_tmp = q.i_*r_ + q.r_*i_ - q.k_*j_ + q.j_*k_;
            fp_type j_tmp = q.j_*r_ + q.k_*i_ + q.r_*j_ - q.i_*k_;
            k_ = q.k_*r_ - q.j_*i_ + q.i_*j_ + q.r_*k_;
            j_ = j_tmp;
            i_ = i_tmp;
            r_ = r_tmp;
        }

        __host__ __device__ void invert_inplace(){
            fp_type ss = sum_of_squares();
            r_ = r_ / ss;
            i_ = -i_ / ss;
            j_ = -j_ / ss;
            k_ = -k_ / ss;
        }

        __host__ __device__ bool is_unit(fp_type tolarance = 1e-14) const{
            return (FABS(1.0) - sum_of_squares()) < tolarance;
        }

        __host__ __device__ fp_type sum_of_squares() const {
            return r_*r_ + i_*i_ + j_*j_ + k_*k_;
        }

        __host__ __device__ void normalise() {
            if(!is_unit()){
                fp_type n = SQRT(sum_of_squares());
                r_ /= n;
                i_ /= n;
                j_ /= n;
                k_ /= n;
            }
        }

        __host__ __device__ fp_type* rot_matrix() {
            normalise();
            calc_rot_matrix();
            return &rot_matrix_[0];
        }

        __host__ __device__ fp_type r() const {return r_;}
        __host__ __device__ fp_type i() const {return i_;}
        __host__ __device__ fp_type j() const {return j_;}
        __host__ __device__ fp_type k() const {return k_;}

        __host__ __device__ void print(char* pre_str){
            printf("%s\nr i j k: %f %f %f %f\n", pre_str, r_, i_, j_, k_);
            calc_rot_matrix();
            printf("Rot matrix:\n%f %f %f\n%f %f %f\n%f %f %f\n",
                    rot_matrix_[0], rot_matrix_[1], rot_matrix_[2],
                    rot_matrix_[3], rot_matrix_[4], rot_matrix_[5],
                    rot_matrix_[6], rot_matrix_[7], rot_matrix_[8]);
        }

        // NOTE, the default copy constructor of this class might cause problems

    private:
        __host__ __device__ void calc_rot_matrix(){
            // calc rotation matrix, doing matrix mult inplace
            fp_type r2 = r_*r_;
            fp_type i2 = i_*i_;
            fp_type j2 = j_*j_;
            fp_type k2 = k_*k_;
            fp_type ij = i_*j_;
            fp_type rk = r_*k_;
            fp_type ik = i_*k_;
            fp_type rj = r_*j_;
            fp_type jk = j_*k_;
            fp_type ri = r_*i_;

            rot_matrix_[0] = i2 + r2 - k2 - j2;
            rot_matrix_[1] = ij - rk - rk + ij;
            rot_matrix_[2] = ik + rj + ik + rj;
            rot_matrix_[3] = ij + rk + rk + ij;
            rot_matrix_[4] = j2 - k2 + r2 - i2;
            rot_matrix_[5] = jk + jk - ri - ri;
            rot_matrix_[6] = ik - rj + ik - rj;
            rot_matrix_[7] = jk + jk + ri + ri;
            rot_matrix_[8] = k2 - j2 - i2 + r2;
        }

        fp_type r_, i_, j_, k_;
        fp_type rot_matrix_[9]; // 3x3
};

class Box{
    public:
        __host__ __device__ Box(fp_type center_x, fp_type center_y, fp_type center_z,
                fp_type size_x, fp_type size_y, fp_type size_z,
                Quaternion &q, fp_type vel_x, fp_type vel_y, fp_type vel_z) :
            cx(center_x), cy(center_y), cz(center_z),
            sx(size_x), sy(size_y), sz(size_z),
            orientation(q), vx(vel_x), vy(vel_y), vz(vel_z) { }

        __host__ __device__ fp_type center_x() const { return cx;}
        __host__ __device__ fp_type center_y() const { return cy;}
        __host__ __device__ fp_type center_z() const { return cz;}
        __host__ __device__ fp_type size_x() const { return sx;}
        __host__ __device__ fp_type size_y() const { return sy;}
        __host__ __device__ fp_type size_z() const { return sz;}
        __host__ __device__ fp_type vel_x() const { return vx;}
        __host__ __device__ fp_type vel_y() const { return vy;}
        __host__ __device__ fp_type vel_z() const { return vz;}
        __host__ __device__ fp_type r() const { return orientation.r(); }
        __host__ __device__ fp_type i() const { return orientation.i(); }
        __host__ __device__ fp_type j() const { return orientation.j(); }
        __host__ __device__ fp_type k() const { return orientation.k(); }

        __host__ __device__ void translate(fp_type x, fp_type y, fp_type z){
            cx += x;
            cy += y;
            cz += z;
        }

        __host__ __device__ void rotate(Quaternion& q){
            fp_type* rm = q.rot_matrix();
            fp_type cx_tmp = rm[0]*cx + rm[1]*cy + rm[2]*cz;
            fp_type cy_tmp = rm[3]*cx + rm[4]*cy + rm[5]*cz;
            cz = rm[6]*cx + rm[7]*cy + rm[8]*cz;
            cy = cy_tmp;
            cx = cx_tmp;

            orientation.rmul_inplace(q);

            fp_type vx_tmp = rm[0]*vx + rm[1]*vy + rm[2]*vz;
            fp_type vy_tmp = rm[3]*vx + rm[4]*vy + rm[5]*vz;
            vz = rm[6]*vx + rm[7]*vy + rm[8]*vz;
            vy = vy_tmp;
            vx = vx_tmp;
        } 

        __host__ __device__ void print(char *pre_str){
            printf("%s\n"
                    "Center:      %f %f %f\n"
                    "Size:        %f %f %f\n"
                    "Velocity:    %f %f %f\n"
                    "Orientation: %f %f %f %f\n",
                    pre_str, cx, cy, cz, sx, sy, sz, vx, vy, vz, r(), i(), j(), k());
        }
    private:
        fp_type cx, cy, cz, sx, sy, sz, vx, vy, vz;
        Quaternion orientation;
};

__global__ void forecast_cuda_kernel(
        const two_dim_pa32<fp_type>   pred_boxes,
        const one_dim_pa32<long>  past_pose_indexes,
        const two_dim_pa32<fp_type>   past_poses,
        const one_dim_pa32<fp_type>   cur_pose,
        const one_dim_pa32<long>      past_ts,
        const long                    cur_ts,
        two_dim_pa32<fp_type>         forecasted_boxes) {
    // blockIdx.x is the block id
    // blockDim.x is the number of threads in a block
    // threadIdx.x is the thread id in the block
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < pred_boxes.size(0)){
        fp_type axis[3] = {0., 0., 1.};
        Quaternion q(axis, pred_boxes[idx][6]);
        Box pred_box(pred_boxes[idx][0], pred_boxes[idx][1], pred_boxes[idx][2],
                pred_boxes[idx][3], pred_boxes[idx][4], pred_boxes[idx][5],
                q, pred_boxes[idx][7], pred_boxes[idx][8], 0);
        auto pose_idx = past_pose_indexes[idx];

        Quaternion csr_q(past_poses[pose_idx][3], past_poses[pose_idx][4],
                past_poses[pose_idx][5], past_poses[pose_idx][6]);
        pred_box.rotate(csr_q);

        pred_box.translate(past_poses[pose_idx][0], past_poses[pose_idx][1],
                past_poses[pose_idx][2]);

        Quaternion epr_q(past_poses[pose_idx][10], past_poses[pose_idx][11],
                past_poses[pose_idx][12], past_poses[pose_idx][13]);
        pred_box.rotate(epr_q);

        pred_box.translate(past_poses[pose_idx][7], past_poses[pose_idx][8],
                past_poses[pose_idx][9]);

        fp_type elapsed_sec = (fp_type)(cur_ts - past_ts[pose_idx]) / 1000000.0;
        fp_type x_diff = pred_box.vel_x()*elapsed_sec;
        fp_type y_diff = pred_box.vel_y()*elapsed_sec;
        if (isfinite(x_diff) && isfinite(y_diff))
            pred_box.translate(x_diff, y_diff, 0);

        // Now use cure pose but inverted
        pred_box.translate(-cur_pose[7], -cur_pose[8], -cur_pose[9]);

        Quaternion epr_inv_q(cur_pose[10], cur_pose[11], cur_pose[12], cur_pose[13]);
        epr_inv_q.invert_inplace();
        pred_box.rotate(epr_inv_q);

        pred_box.translate(-cur_pose[0], -cur_pose[1], -cur_pose[2]);

        Quaternion csr_inv_q(cur_pose[3], cur_pose[4], cur_pose[5], cur_pose[6]);
        csr_inv_q.invert_inplace();
        pred_box.rotate(csr_inv_q);

        forecasted_boxes[idx][0] = pred_box.center_x();
        forecasted_boxes[idx][1] = pred_box.center_y();
        forecasted_boxes[idx][2] = pred_box.center_z();
        forecasted_boxes[idx][3] = pred_box.size_x();
        forecasted_boxes[idx][4] = pred_box.size_y();
        forecasted_boxes[idx][5] = pred_box.size_z();

        fp_type r = pred_box.r();
        fp_type i = pred_box.i();
        fp_type j = pred_box.j();
        fp_type k = pred_box.k();

        forecasted_boxes[idx][6] = 2 * ATAN2(SQRT(i*i+j*j+k*k), r);
        forecasted_boxes[idx][7] = pred_box.vel_x();
        forecasted_boxes[idx][8] = pred_box.vel_y();
    }
}

void forecast_cpu_kernel(
        const two_dim_acc<fp_type>   pred_boxes,
        const one_dim_acc<long>      past_pose_indexes,
        const two_dim_acc<fp_type>   past_poses,
        const one_dim_acc<fp_type>   cur_pose,
        const one_dim_acc<long>      past_ts,
        const long                   cur_ts,
        two_dim_acc<fp_type>        forecasted_boxes) {
    // blockIdx.x is the block id
    // blockDim.x is the number of threads in a block
    // threadIdx.x is the thread id in the block
    for(auto idx=0; idx < pred_boxes.size(0); ++idx){
        fp_type axis[3] = {0., 0., 1.};
        Quaternion q(axis, pred_boxes[idx][6]);
        Box pred_box(pred_boxes[idx][0], pred_boxes[idx][1], pred_boxes[idx][2],
                pred_boxes[idx][3], pred_boxes[idx][4], pred_boxes[idx][5],
                q, pred_boxes[idx][7], pred_boxes[idx][8], 0);
        auto pose_idx = past_pose_indexes[idx];

        Quaternion csr_q(past_poses[pose_idx][3], past_poses[pose_idx][4],
                past_poses[pose_idx][5], past_poses[pose_idx][6]);
        pred_box.rotate(csr_q);

        pred_box.translate(past_poses[pose_idx][0], past_poses[pose_idx][1],
                past_poses[pose_idx][2]);

        Quaternion epr_q(past_poses[pose_idx][10], past_poses[pose_idx][11],
                past_poses[pose_idx][12], past_poses[pose_idx][13]);
        pred_box.rotate(epr_q);

        pred_box.translate(past_poses[pose_idx][7], past_poses[pose_idx][8],
                past_poses[pose_idx][9]);

        fp_type elapsed_sec = (fp_type)(cur_ts - past_ts[pose_idx]) / 1000000.0;
        fp_type x_diff = pred_box.vel_x()*elapsed_sec;
        fp_type y_diff = pred_box.vel_y()*elapsed_sec;
        if (isfinite(x_diff) && isfinite(y_diff))
            pred_box.translate(x_diff, y_diff, 0);

        // Now use cure pose but inverted
        pred_box.translate(-cur_pose[7], -cur_pose[8], -cur_pose[9]);

        Quaternion epr_inv_q(cur_pose[10], cur_pose[11], cur_pose[12], cur_pose[13]);
        epr_inv_q.invert_inplace();
        pred_box.rotate(epr_inv_q);

        pred_box.translate(-cur_pose[0], -cur_pose[1], -cur_pose[2]);

        Quaternion csr_inv_q(cur_pose[3], cur_pose[4], cur_pose[5], cur_pose[6]);
        csr_inv_q.invert_inplace();
        pred_box.rotate(csr_inv_q);

        forecasted_boxes[idx][0] = pred_box.center_x();
        forecasted_boxes[idx][1] = pred_box.center_y();
        forecasted_boxes[idx][2] = pred_box.center_z();
        forecasted_boxes[idx][3] = pred_box.size_x();
        forecasted_boxes[idx][4] = pred_box.size_y();
        forecasted_boxes[idx][5] = pred_box.size_z();

        fp_type r = pred_box.r();
        fp_type i = pred_box.i();
        fp_type j = pred_box.j();
        fp_type k = pred_box.k();

        forecasted_boxes[idx][6] = 2 * ATAN2(SQRT(i*i+j*j+k*k), r);
        forecasted_boxes[idx][7] = pred_box.vel_x();
        forecasted_boxes[idx][8] = pred_box.vel_y();
    }
}


// This will be on cpu
void move_to_world_coords_kernel(
        const two_dim_acc<fp_type> pred_boxes,
        const two_dim_acc<fp_type> poses,
        const one_dim_acc<long>    pose_idx,
        two_dim_acc<fp_type>       moved_boxes) {

    for(auto idx=0; idx < pred_boxes.size(0); ++idx){
        fp_type axis[3] = {0., 0., 1.};
        Quaternion q(axis, pred_boxes[idx][6]);
        Box pred_box(pred_boxes[idx][0], pred_boxes[idx][1], pred_boxes[idx][2],
                pred_boxes[idx][3], pred_boxes[idx][4], pred_boxes[idx][5],
                q, pred_boxes[idx][7], pred_boxes[idx][8], 0);

        auto pose_idx_ = pose_idx[idx];

        Quaternion csr_q(poses[pose_idx_][3], poses[pose_idx_][4], poses[pose_idx_][5], poses[pose_idx_][6]);
        pred_box.rotate(csr_q);

        pred_box.translate(poses[pose_idx_][0], poses[pose_idx_][1],poses[pose_idx_][2]);

        Quaternion epr_q(poses[pose_idx_][10], poses[pose_idx_][11],poses[pose_idx_][12], poses[pose_idx_][13]);
        pred_box.rotate(epr_q);

        pred_box.translate(poses[pose_idx_][7], poses[pose_idx_][8],poses[pose_idx_][9]);

        moved_boxes[idx][0] = pred_box.center_x();
        moved_boxes[idx][1] = pred_box.center_y();
        moved_boxes[idx][2] = pred_box.center_z();
        moved_boxes[idx][3] = pred_box.size_x();
        moved_boxes[idx][4] = pred_box.size_y();
        moved_boxes[idx][5] = pred_box.size_z();

        fp_type r = pred_box.r();
        fp_type i = pred_box.i();
        fp_type j = pred_box.j();
        fp_type k = pred_box.k();

        moved_boxes[idx][6] = 2 * ATAN2(SQRT(i*i+j*j+k*k), r);
        moved_boxes[idx][7] = pred_box.vel_x();
        moved_boxes[idx][8] = pred_box.vel_y();
    }
}

torch::Tensor forecast_past_dets(
        const torch::Tensor pred_boxes, // [num_objects, 9], fp_type
        const torch::Tensor past_pose_indexes, // [num_objects], long
        const torch::Tensor past_poses, // [num_past_poses, 14], fp_type
        const torch::Tensor cur_pose, // [14], fp_type
        const torch::Tensor past_timestamps, // [num_past_poses], long
        const long target_timestamp // [1]
        )
{
    torch::Tensor forecasted_boxes = torch::empty_like(pred_boxes);

    if (pred_boxes.device().type() == torch::kCPU){
        forecast_cpu_kernel(
                pred_boxes.accessor<fp_type,2>(),
                past_pose_indexes.accessor<long,1>(),
                past_poses.accessor<fp_type,2>(),
                cur_pose.accessor<fp_type,1>(),
                past_timestamps.accessor<long,1>(),
                target_timestamp,
                forecasted_boxes.accessor<fp_type,2>());
    }
    else{
        const auto threads_per_block = 256;
        const auto num_blocks = std::ceil((fp_type)pred_boxes.size(0) / threads_per_block);
        auto s = at::cuda::getCurrentCUDAStream();
        forecast_cuda_kernel<<<num_blocks, threads_per_block, 0, s>>>(
                pred_boxes.packed_accessor32<fp_type,2,torch::RestrictPtrTraits>(),
                past_pose_indexes.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
                past_poses.packed_accessor32<fp_type,2,torch::RestrictPtrTraits>(),
                cur_pose.packed_accessor32<fp_type,1,torch::RestrictPtrTraits>(),
                past_timestamps.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
                target_timestamp,
                forecasted_boxes.packed_accessor32<fp_type,2,torch::RestrictPtrTraits>());
    }

    return forecasted_boxes;
}


torch::Tensor move_to_world_coords(
        const torch::Tensor pred_boxes, // [num_objects, 9], fp_type
        const torch::Tensor poses, // [num_objects, 9], fp_type
        const torch::Tensor pose_idx // [14], fp_type
        )
{
    torch::Tensor moved_boxes = torch::empty_like(pred_boxes);

    move_to_world_coords_kernel(pred_boxes.accessor<fp_type, 2>(),
            poses.accessor<fp_type, 2>(),
            pose_idx.accessor<long, 1>(),
            moved_boxes.accessor<fp_type, 2>());

    return moved_boxes;
}

// hp: high priority, lp: low priority
torch::Tensor forecasting_nms(
        const torch::Tensor pred_boxes_hp,
        const torch::Tensor pred_labels_hp,
        const torch::Tensor pred_boxes_lp,
        const torch::Tensor pred_labels_lp,
        const double iou_threshold)
{
    using namespace torch::indexing;

    auto tensor_options = torch::TensorOptions()
        .dtype(torch::kFloat32)
        .device(torch::kCPU)
        .requires_grad(false);

    torch::Tensor ans_iou = torch::zeros({pred_boxes_hp.size(0), pred_boxes_lp.size(0)},
            tensor_options);
    auto pb_hp = pred_boxes_hp.index({Slice(), Slice(None, 7)}).contiguous();
    auto pb_lp = pred_boxes_lp.index({Slice(), Slice(None, 7)}).contiguous();
    boxes_iou_bev_with_labels_cpu(pb_hp, pred_labels_hp, pb_lp, pred_labels_lp, ans_iou);

    torch::Tensor keep_mask = torch::ones(pred_boxes_lp.size(0), tensor_options.dtype(torch::kBool));

    auto iou_a = ans_iou.accessor<float, 2>();
    auto lbl_hp = pred_labels_hp.accessor<long, 1>();
    auto lbl_lp = pred_labels_lp.accessor<long, 1>();
    
    for(auto i=0; i<iou_a.size(0); ++i){
        for(auto j=0; j<iou_a.size(1); ++j){
            if(iou_a[i][j] >= iou_threshold){
                keep_mask[j] = false;
            }
        }
    }
    
    return keep_mask;
}


std::vector<std::map<std::string, torch::Tensor>> split_dets(
        const torch::Tensor pred_boxes, // [num_objects, 9], fp_type
        const torch::Tensor pred_scores,
        const torch::Tensor pred_labels,
        const torch::Tensor cls_id_to_det_head_idx_map,
        const long num_det_heads,
        const bool move_to_gpu)
{
    using namespace torch::indexing;
    torch::Tensor det_head_mappings = cls_id_to_det_head_idx_map.index({pred_labels});

    std::vector<std::map<std::string, torch::Tensor>> forc_dicts(num_det_heads);
    auto pred_merged = torch::cat({pred_boxes, pred_scores.unsqueeze(-1),
            pred_labels.to(torch::kFloat32).unsqueeze(-1)}, -1);
    for(auto i=0; i<num_det_heads; ++i){
        std::map<std::string, torch::Tensor> pdict;

        auto pred_masked = pred_merged.index({(det_head_mappings == i)});
        auto pb = pred_masked.index({Slice(), Slice(None, -2)});
        auto ps = pred_masked.index({Slice(), -2});
        auto pl = pred_masked.index({Slice(), -1}).to(torch::kLong);
        pdict["pred_boxes"] = (move_to_gpu ? pb.to(torch::kCUDA) : pb);
        pdict["pred_scores"] = (move_to_gpu ? ps.to(torch::kCUDA) : ps);
        pdict["pred_labels"] = (move_to_gpu ? pl.to(torch::kCUDA) : pl);
        forc_dicts[i] = std::move(pdict);
    }
    return forc_dicts;
}
